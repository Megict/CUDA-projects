#include "hip/hip_runtime.h"
#include <thrust/extrema.h>
#include <thrust/device_vector.h>
#include <hip/hip_fp16.h>

#include <iostream>
#include <vector>
#include <string>
#include <fstream>
#include <bitset>
#include <chrono>

#define silent false 	//no stderr except actual errors
#define verbal true		 
#define visual false		 
#define debug  false	//do printf in kernel

#define gridsize 32

#define INDEX_ERROR 800

#define EPS 1e-7

#define CSC(call)  																											\
do {																														\
	hipError_t err = call;																									\
	if (err != hipSuccess) {																								\
		std::cerr << "ERROR in " << __FILE__ << ":" << __LINE__ << ". Message: " << hipGetErrorString(err) << "\n";		\
		exit(0);																											\
	}																														\
} while(0)

//вариант 6
//Нахождение ранга матрицы

__global__ void kernel_swap_rows(double* elements, int row_1, int row_2, int n, int m) {
	int idx = blockIdx.x * blockDim.x + threadIdx.x;
	int offset_x = blockDim.x * gridDim.x;

	double swp;
	while(idx < m) {
		swp = elements[row_1*m + idx];
		elements[row_1*m + idx] = elements[row_2*m + idx];
		elements[row_2*m + idx] = swp;
		
		if (debug) printf("KERNEL: swaping %lf and %lf at idx %d\n",elements[row_1*m + idx], elements[row_2*m + idx], idx);

		idx += offset_x;
	}
}

__device__ bool dev_close_to_zero(double val) {
	if(fabs(val) < EPS) {
		return true;
	}
	else {
		return false;
	}
} 

__global__ void kernel_gaussian_step(double* elements, int n, int m, int start_row_index, int active_colomn) {
	//n - количество строк (элементов в столбце)
	//m - количество столбцов (элементов в строке)

	int idx = blockIdx.x * blockDim.x + threadIdx.x;
	int idy = blockIdx.y * blockDim.y + threadIdx.y;

	int cur_row = idy + start_row_index + 1;
	int cur_col = idx + active_colomn + 1; 
	int in_row_offset = blockDim.x * gridDim.x; 
	int other_row_offset = blockDim.y * gridDim.y; 

	__shared__ double coef[32][32];

	while(cur_row < n) {
		
		coef[blockIdx.x][blockIdx.y] = - elements[cur_row*m + active_colomn] / elements[start_row_index*m + active_colomn];

		while(cur_col < m) {
			elements[cur_row*m + cur_col] = elements[cur_row*m + cur_col] + coef[blockIdx.x][blockIdx.y]*elements[start_row_index*m + cur_col];
			cur_col += in_row_offset;
		}
		cur_row += other_row_offset;
		cur_col = idx + active_colomn + 1; 
	}
}

__global__ void kernel_extract_col(double* elements, double* target, int n, int m, int start_row_index, int active_colomn) {
	int idx = start_row_index + blockIdx.x * blockDim.x + threadIdx.x;
	int offset_x = blockDim.x * gridDim.x;

	while(idx < n) {
		target[idx - start_row_index] = elements[idx*m + active_colomn];

		idx += offset_x;
	}

}


bool close_to_zero(double val) {
	if(fabs(val) < EPS) {
		return true;
	}
	return false;
}



struct Compare {
    __host__ __device__ bool operator()(double num1, double num2) {
        return fabs(num1) < fabs(num2);
    }
};


class matrix{
	int n;
	int m;
	double* array;
	double* device_matrix;
	
	double* max_elm_val; //нужно для одной функции
	double* dev_tmp_vec;

	

	Compare cmp;
	thrust::device_ptr<double> device_ptr;

public:
	matrix(int n_, int m_, double* array_) {
		n = n_;
		m = m_;
		array = array_;
		CSC(hipMalloc (&device_matrix, sizeof(double)*m*n));
		CSC(hipMemcpy (device_matrix, array, sizeof(double)*m*n, hipMemcpyHostToDevice));
		CSC(hipMalloc (&dev_tmp_vec, sizeof(double)*n));
		device_ptr = thrust::device_pointer_cast(dev_tmp_vec);
		

		max_elm_val = (double*)malloc(sizeof(double));
	}

	matrix(int n_, int m_){ //считывание матрицы с stdin
		n = n_; m = m_;
		//n - количество строк (элементов в столбце)
		//m - количество столбцов (элементов в строке)
		double* arr_all = (double*)malloc(sizeof(double)*m*n);

		for (int i = 0; i < n; ++i){ //проход по строкам
			for (int j = 0; j < m; ++j){ //проход по столбцам
				double elm = 0;
				std::cin >> elm;
				arr_all[i*m + j] = elm;
			}
		}

		array = arr_all;

		CSC(hipMalloc (&device_matrix, sizeof(double)*m*n));
		CSC(hipMemcpy (device_matrix, array, sizeof(double)*m*n, hipMemcpyHostToDevice));
		CSC(hipMalloc (&dev_tmp_vec, sizeof(double)*n));
		device_ptr = thrust::device_pointer_cast(dev_tmp_vec);

		max_elm_val = (double*)malloc(sizeof(double));
	}

	~matrix() {
		free(array);
		free(max_elm_val);
		CSC(hipFree(device_matrix));
	}

	void update_host_matrix() {
		CSC(hipMemcpy (array, device_matrix, sizeof(double)*m*n, hipMemcpyDeviceToHost));
	}

	void update_device_matrix() {
		CSC(hipMemcpy (device_matrix, array, sizeof(double)*m*n, hipMemcpyHostToDevice));
	}

	void print() {

		for (int i = 0; i < n; ++i) {
			for (int j = 0; j < m; ++j) {
				std::cerr << array[i*m + j] << " ";
			}
			
			std::cerr << "\n";
		}
	}

	void printf() {

		for (int i = 0; i < n; ++i) {
			for (int j = 0; j < m; ++j) {
				
				if(array[i*m + j] >= 0) {
					std::printf(" ");
				}
				std::printf("%.2lf ",array[i*m + j]);

			}
			std::printf("\n");
		}
	}

	int find_max_elm(int start_row, int start_col,int len) {
		//находит максимальный элемент массива и возвращает его индекс
		//если максимальный элемент = 0, то возвращает -1
		if(visual) {
			std::cerr << "\tmax elm search | start " << start_row << " , " << start_col << " end " << start_row + len << " , " << start_col << "\n";
		}

		kernel_extract_col<<<256,256>>>(device_matrix, dev_tmp_vec + start_row, n, m, start_row, start_col);

		int max_elm_pos = thrust::max_element(device_ptr + start_row, device_ptr + start_row + len, cmp) - device_ptr - start_row;
		
		//CSC(hipMemcpy (max_elm_val, &device_matrix[(array_start / n + max_elm_pos) * n + array_start % n], sizeof(double), hipMemcpyDeviceToHost));
		CSC(hipMemcpy (max_elm_val, &dev_tmp_vec[start_row + max_elm_pos], sizeof(double), hipMemcpyDeviceToHost));
		
		if(visual) {
			std::cerr << "\tmax elm val is " << max_elm_val[0] << "\n";
		}

		if(close_to_zero(max_elm_val[0])) {
			return -1;
		}

		return max_elm_pos;
	}

	void swap_rows(int lhs,int rhs) {
		//меняет местами строки lhs и rhs

		if(lhs >= n || rhs >= n) {
			throw INDEX_ERROR;
		}

		if(lhs == rhs) {
			return;
		}

		kernel_swap_rows<<<256,256>>>(device_matrix, lhs, rhs, n, m);

	}

	//1 - определить ведущий элемент в столбце i (thrust)
	//2 - переставить строки местами (О(m)) (параллельно, одномерной сеткой)
	//3 - вычислить коэфециенты для каждой строки (O(n)) ДЕЛАЕТСЯ ВНУТРИ ЯДРА
	//4 - записать коэфициенты в разд. память. 
	//		Каждый блок работает с одной из строк, ему нужен только один коэфициент
	//		Первый варп каждого блока считает коэфициент для соответствующей строки, следующие вары извлекают его из разделяемой памяти
	//		У каждого блока должен быть поток-лидер, который помещает нужный элемент в разделяемую память, остальные потоки должны начать работу только после завершения перемещения
	//		варп потока-лидера будет работать неоптимально
	//5 - преобразовать строки (O(n*n)) (параллельно, двумерной сеткой)
	
	int rank() {
		hipEvent_t start, stop;
		float time_summary = 0;
		float time_max = 0;
		float time_swap = 0;
		float milliseconds = 0;
		CSC(hipEventCreate(&start));
		CSC(hipEventCreate(&stop));
		std::chrono::steady_clock::time_point max_begin;
		std::chrono::steady_clock::time_point max_end;
		float max_time = 0;

		int rank = -1;
		int active_element_idx = -1;

		std::chrono::steady_clock::time_point cc_begin = std::chrono::steady_clock::now();
		for (int i = 0; i < n - 1; ++i) { //i - текущая строка.
		
			active_element_idx += 1;

			if(active_element_idx == m) {
				rank = i;
				break;
			}


			CSC(hipEventRecord(start));
			max_begin = std::chrono::steady_clock::now();
			int max_elm_idx = find_max_elm(i, active_element_idx, n - i) + i;

			while(max_elm_idx - i == -1 && active_element_idx + 1 < m) {//максимальный элемент строки равен нулю
				active_element_idx += 1;
				int max_elm_idx = find_max_elm(i, active_element_idx, n - i) + i;
			}
			max_end = std::chrono::steady_clock::now();

			CSC(hipEventRecord(stop));
			CSC(hipEventSynchronize(stop));
			milliseconds = 0;
			CSC(hipEventElapsedTime(&milliseconds, start, stop));
			time_max += milliseconds;
			max_time += std::chrono::duration_cast<std::chrono::microseconds>(max_end - max_begin).count();

			if(active_element_idx + 1 == m && max_elm_idx - i == -1) {
				rank = i;
				break;
			}


			if(i != max_elm_idx) {
				CSC(hipEventRecord(start));
				swap_rows(i,max_elm_idx);
				CSC(hipEventRecord(stop));
				CSC(hipEventSynchronize(stop));
				milliseconds = 0;
				CSC(hipEventElapsedTime(&milliseconds, start, stop));
				time_swap += milliseconds;
			}
									
			CSC(hipEventRecord(start));
			kernel_gaussian_step<<<dim3(gridsize,gridsize),dim3(gridsize,gridsize)>>> (device_matrix, n, m, i, active_element_idx);
			CSC(hipEventRecord(stop));
			CSC(hipEventSynchronize(stop));
			milliseconds = 0;
			CSC(hipEventElapsedTime(&milliseconds, start, stop));
			time_summary += milliseconds;


			if(active_element_idx + 1 == m) {
				rank = i + 1;
				break;
			}
		}
		std::chrono::steady_clock::time_point cc_end = std::chrono::steady_clock::now();
		std::cerr << "Finished cycle\n";
		std::cout << "total cc time: " << std::chrono::duration_cast<std::chrono::microseconds>(cc_end - cc_begin).count() / 1000 << " ms \n";
		std::cout << "total max cpu time: " << max_time / 1000 << " ms \n";

		update_host_matrix();
		//после завершения цикла нужно определить, занулилась ли последняя строка и, тем самым, понять, каков ранг
		//если цикл завершился до последней строки, то ранг уже вычислен
		if(rank == -1) {
			//надо определить, есть ли среди последних m - n элементов ненулевые
			rank = n - 1;
			for (int i = n - 1; i < m; ++i) { //было n-1
				std::cerr << array[(n-1)*m + i] << "\n";
				if(!close_to_zero(array[(n-1)*m + i])) {
					rank = n;
					break;
				}
			}
		}

		if (verbal) {
			std::cerr << rank << "\n";
			//printf();
			std::cerr << "total main kernel time: " << time_summary << " ms (" << time_summary / 1000 << " s)\n";
			std::cerr << "total max kernel time: " << time_max << " ms (" << time_max / 1000 << " s)\n";
			std::cerr << "total swap kernel time: " << time_swap << " ms (" << time_swap / 1000 << " s)\n";
		}

		return rank;

	}

};

int main() {
	
	try{ 
		int n,m;
		//std::cin >> n >> m;
		n = 2000;
		m = 10000;
		std::chrono::steady_clock::time_point begin = std::chrono::steady_clock::now();

		double* arr = (double*)malloc(m*n*sizeof(double));
		for(int i = 0; i < n*m; ++i) {
			arr[i] = std::rand()%100;
		}

		matrix matr(n,m,arr);

		std::cerr << "shape: "<< n << " " << m << "\n";

		if(visual) {
			std::cerr << "--\n";
			matr.print();
			std::cerr << "\n";
			matr.printf();
			std::cerr << "\n";
		}

		std::chrono::steady_clock::time_point rank_begin = std::chrono::steady_clock::now();
		int rank = matr.rank();
		std::chrono::steady_clock::time_point rank_end = std::chrono::steady_clock::now();

		if(visual) {
			std::cerr << "-- RANK: " << rank << " --\n";
		}
		std::cout.precision(10);
		std::cout << rank << std::fixed << "\n";
		
		std::chrono::steady_clock::time_point end = std::chrono::steady_clock::now();
		std::cout << "total cpu time: " << std::chrono::duration_cast<std::chrono::microseconds>(end - begin).count() / 1000 << " ms \n";
		std::cout << "rank calc cpu time: " << std::chrono::duration_cast<std::chrono::microseconds>(rank_end - rank_begin).count() / 1000 << " ms \n";
	}
	catch(int err) {
		if (err == 101) {
			std::cerr << "error opening file\n";
		} else
		if (err == 105){
			std::cerr << "error new length\n";
		} else 
		if (err == 800) {
			std::cerr << "error index\n";
		} else{
			std::cerr << "unknown error detected\n";
		}
	}

	return 0;
}


