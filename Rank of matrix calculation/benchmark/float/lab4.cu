#include "hip/hip_runtime.h"
#include <thrust/extrema.h>
#include <thrust/device_vector.h>
#include <hip/hip_fp16.h>

#include <iostream>
#include <vector>
#include <string>
#include <fstream>
#include <bitset>
#include <chrono>

#define silent false 	//no stderr except actual errors
#define verbal true		 
#define visual false		 
#define debug  false	//do printf in kernel

#define gridsize 32

#define INDEX_ERROR 800

#define EPS 1e-7

#define CSC(call)  																											\
do {																														\
	hipError_t err = call;																									\
	if (err != hipSuccess) {																								\
		std::cerr << "ERROR in " << __FILE__ << ":" << __LINE__ << ". Message: " << hipGetErrorString(err) << "\n";		\
		exit(0);																											\
	}																														\
} while(0)

//вариант 6
//Нахождение ранга матрицы

__global__ void kernel_swap_rows(float* elements, int row_1, int row_2, int n, int m) {
	int idx = blockIdx.x * blockDim.x + threadIdx.x;
	int offset_x = blockDim.x * gridDim.x;

	float swp;
	while(idx < m) {
		swp = elements[idx*n + row_1];
		elements[idx*n + row_1] = elements[idx*n + row_2];
		elements[idx*n + row_2] = swp;
		
		if (debug) printf("KERNEL: swaping %f and %f at idx %d\n",elements[idx*n + row_1], elements[idx*n + row_2], idx);

		idx += offset_x;
	}
}

__device__ bool dev_close_to_zero(float val) {
	if(val < EPS && val > -EPS) {
		return true;
	}
	else {
		return false;
	}
} 

__global__ void kernel_gaussian_step(float* elements, int n, int m, int start_row_index, int active_colomn) {
	//n - количество строк (элементов в столбце)
	//m - количество столбцов (элементов в строке)
	if(dev_close_to_zero(elements[active_colomn*n + start_row_index] )){
		return;
	}

	int idx = blockIdx.x * blockDim.x + threadIdx.x;
	int idy = blockIdx.y * blockDim.y + threadIdx.y;

	int cur_row = idx + start_row_index + 1;
	int cur_col = idy + active_colomn + 1; 
	int in_row_offset = blockDim.y * gridDim.y;
	int other_row_offset = blockDim.x * gridDim.x; 

	__shared__ float coef[32];

	while(cur_row < n) {
		
		coef[threadIdx.x] = - elements[active_colomn*n + cur_row] / elements[active_colomn*n + start_row_index];

		while(cur_col < m) {

			elements[cur_col*n + cur_row] = elements[cur_col*n + cur_row] + coef[threadIdx.x]*elements[cur_col*n + start_row_index];
			cur_col += in_row_offset;
		}
		cur_row += other_row_offset;
		cur_col = idy + active_colomn + 1;
	}
}

bool close_to_zero(float val) {
	if(val < EPS && val > -EPS) {
		return true;
	}
	return false;
}



struct Compare {
    __host__ __device__ bool operator()(float num1, float num2) {
        return fabs(num1) < fabs(num2);
    }
};


class matrix{
	int n;
	int m;
	float* array;
	float* device_matrix;
	
	float* max_elm_val; //нужно для одной функции

public:
	matrix(int n_, int m_, float* array_) {
		n = n_;
		m = m_;
		array = array_;
		CSC(hipMalloc (&device_matrix, sizeof(float)*m*n));
		CSC(hipMemcpy (device_matrix, array, sizeof(float)*m*n, hipMemcpyHostToDevice));
						
		max_elm_val = (float*)malloc(sizeof(float));
	}

	matrix(int n_, int m_){ //считывание матрицы с stdin
		n = n_; m = m_;
		//n - количество строк (элементов в столбце)
		//m - количество столбцов (элементов в строке)
		float* arr_all = (float*)malloc(sizeof(float)*m*n);

		for (int i = 0; i < n; ++i){ //проход по строкам
			for (int j = 0; j < m; ++j){ //проход по столбцам
				float elm = 0;
				std::cin >> elm;
				arr_all[j*n + i] = elm;
			}
		}

		array = arr_all;

		CSC(hipMalloc (&device_matrix, sizeof(float)*m*n));
		CSC(hipMemcpy (device_matrix, array, sizeof(float)*m*n, hipMemcpyHostToDevice));

		max_elm_val = (float*)malloc(sizeof(float));
	}

	~matrix() {
		free(array);
		free(max_elm_val);
		CSC(hipFree(device_matrix));
	}

	void update_host_matrix() {
		CSC(hipMemcpy (array, device_matrix, sizeof(float)*m*n, hipMemcpyDeviceToHost));
	}

	void update_device_matrix() {
		CSC(hipMemcpy (device_matrix, array, sizeof(float)*m*n, hipMemcpyHostToDevice));
	}

	void print() {

		for (int i = 0; i < m; ++i) {
			for (int j = 0; j < n; ++j) {
				std::cerr << array[i*n + j] << " ";
			}
			//std::cerr << "\n---\n";
			std::cerr << "\n";
		}
	}

	void printf() {

		for (int j = 0; j < n; ++j) {
			for (int i = 0; i < m; ++i) {
				
				if(array[i*n + j] >= 0) {
					std::printf(" ");
				}
				std::printf("%.2lf ",array[i*n + j]);

			}
			std::printf("\n");
		}
	}

	int find_max_elm(float* device_array,int array_size) {
		//находит максимальный элемент массива и возвращает его индекс
		//если максимальный элемент = 0, то возвращает -1

		Compare cmp;

		thrust::device_ptr<float> device_ptr = thrust::device_pointer_cast(device_array);	
		int max_elm_pos = thrust::max_element(device_ptr, device_ptr + array_size, cmp) - device_ptr;
		
		CSC(hipMemcpy (max_elm_val, &device_array[max_elm_pos], sizeof(float), hipMemcpyDeviceToHost));

		if(close_to_zero(max_elm_val[0])) {
			max_elm_pos = -1;
		}
		
		return max_elm_pos;
	}

	void swap_rows(int lhs,int rhs) {
		//меняет местами строки lhs и rhs

		if(lhs >= n || rhs >= n) {
			throw INDEX_ERROR;
		}

		if(lhs == rhs) {
			return;
		}

		kernel_swap_rows<<<1024,1024>>>(device_matrix, lhs, rhs, n, m);

	}

	//1 - определить ведущий элемент в столбце i (thrust)
	//2 - переставить строки местами (О(m)) (параллельно, одномерной сеткой)
	//3 - вычислить коэфециенты для каждой строки (O(n)) ДЕЛАЕТСЯ ВНУТРИ ЯДРА
	//4 - записать коэфициенты в разд. память. 
	//		Каждый блок работает с одной из строк, ему нужен только один коэфициент
	//		Первый варп каждого блока считает коэфициент для соответствующей строки, следующие вары извлекают его из разделяемой памяти
	//		У каждого блока должен быть поток-лидер, который помещает нужный элемент в разделяемую память, остальные потоки должны начать работу только после завершения перемещения
	//		варп потока-лидера будет работать неоптимально
	//5 - преобразовать строки (O(n*n)) (параллельно, двумерной сеткой)

	
	int rank() {
		hipEvent_t start, stop;
		float time_summary = 0;
		float time_max = 0;
		float time_swap = 0;
		float milliseconds = 0;
		CSC(hipEventCreate(&start));
		CSC(hipEventCreate(&stop));

		int rank = -1;
		int active_element_idx = -1;

		for (int i = 0; i < n - 1; ++i) { //i - текущая строка.
		
			if (visual) {
				std::cerr << "Starting iteration " << i << " of " << n - 1 << "\n";
			}

			active_element_idx += 1;
			if(active_element_idx == m) {
				rank = i;
				break;
			}


			CSC(hipEventRecord(start));
			int max_elm_idx = find_max_elm(&device_matrix[active_element_idx*n + i],n - i) + i;
			CSC(hipEventRecord(stop));
			CSC(hipEventSynchronize(stop));
			milliseconds = 0;
			CSC(hipEventElapsedTime(&milliseconds, start, stop));
			time_max += milliseconds;

			if (visual) {
				std::cerr << "active colomn idx is "<< active_element_idx << " max elm pos is " << max_elm_idx << "\n";
			}

			while(max_elm_idx - i == -1 && active_element_idx + 1 < m) {//максимальный элемент строки равен нулю
				active_element_idx += 1;
				max_elm_idx = find_max_elm(&device_matrix[active_element_idx*n + i],n - i) + i;
				
				if (visual) {
					std::cerr << "active colomn idx is "<< active_element_idx << " max elm pos is " << max_elm_idx << "\n";
				}
			}

			if(active_element_idx + 1 == m && max_elm_idx - i == -1) {
				rank = i;
				break;
			}


			if (visual) {
				std::cerr << "\tcur index " << i << "\tindex with max elm " << max_elm_idx <<"\n";
			}
			if(i != max_elm_idx) {
				CSC(hipEventRecord(start));
				swap_rows(i,max_elm_idx);
				CSC(hipEventRecord(stop));
				CSC(hipEventSynchronize(stop));
				milliseconds = 0;
				CSC(hipEventElapsedTime(&milliseconds, start, stop));
				time_swap += milliseconds;
			}
									
			CSC(hipEventRecord(start));
			kernel_gaussian_step<<<dim3(gridsize,gridsize),dim3(gridsize,gridsize)>>> (device_matrix, n, m, i, active_element_idx);
			CSC(hipEventRecord(stop));
			CSC(hipEventSynchronize(stop));
			milliseconds = 0;
			CSC(hipEventElapsedTime(&milliseconds, start, stop));
			time_summary += milliseconds;

			if (visual) {
				update_host_matrix(); //нужно для вывода, для быстрой работы отключить
				std::cerr << "after transformation\n";
				printf();
			}

			if(active_element_idx + 1 == m) {
				rank = i + 1;
				break;
			}
		}
		std::cerr << "Finished cycle\n";
		update_host_matrix();
		//после завершения цикла нужно определить, занулилась ли последняя строка и, тем самым, понять, каков ранг
		//если цикл завершился до последней строки, то ранг уже вычислен
		if(rank == -1) {
			//надо определить, есть ли среди последних m - n элементов ненулевые
			rank = n - 1;
			for (int i = n - 1; i < m; ++i) { //было n-1
				std::cerr << array[i*n + (n-1)] << "\n";
				if(!close_to_zero(array[i*n + (n-1)])) {
					rank = n;
					break;
				}
			}
		}

		if (verbal) {
			std::cerr << rank << "\n";
			//printf();
			std::cerr << "total main kernel time: " << time_summary << " ms (" << time_summary / 1000 << " s)\n";
			std::cerr << "total max kernel time: " << time_max << " ms (" << time_max / 1000 << " s)\n";
			std::cerr << "total swap kernel time: " << time_swap << " ms (" << time_swap / 1000 << " s)\n";
		}
		
		return rank;

	}

};

int main() {
	
	try{ 
		int n,m;
		//std::cin >> n >> m;
		n = 2000;
		m = 10000;
		std::chrono::steady_clock::time_point begin = std::chrono::steady_clock::now();

		float* arr = (float*)malloc(m*n*sizeof(float));
		for(int i = 0; i < n*m; ++i) {
			arr[i] = std::rand()%100;
		}

		matrix matr(n,m,arr);

		std::cerr << "shape: "<< n << " " << m << "\n";

		if(visual) {
			std::cerr << "--\n";
			matr.print();
			std::cerr << "\n";
			matr.printf();
			std::cerr << "\n";
		}

		std::chrono::steady_clock::time_point rank_begin = std::chrono::steady_clock::now();
		int rank = matr.rank();
		std::chrono::steady_clock::time_point rank_end = std::chrono::steady_clock::now();

		if(visual) {
			std::cerr << "-- RANK: " << rank << " --\n";
		}
		std::cout.precision(10);
		std::cout << rank << std::fixed << "\n";
		
		std::chrono::steady_clock::time_point end = std::chrono::steady_clock::now();
		std::cout << "total cpu time: " << std::chrono::duration_cast<std::chrono::microseconds>(end - begin).count() / 1000 << " ms \n";
		std::cout << "rank calc cpu time: " << std::chrono::duration_cast<std::chrono::microseconds>(rank_end - rank_begin).count() / 1000 << " ms \n";
	}
	catch(int err) {
		if (err == 101) {
			std::cerr << "error opening file\n";
		} else
		if (err == 105){
			std::cerr << "error new length\n";
		} else 
		if (err == 800) {
			std::cerr << "error index\n";
		} else{
			std::cerr << "unknown error detected\n";
		}
	}

	return 0;
}


